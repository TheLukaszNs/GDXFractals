
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

typedef struct {
    float r;
    float i;
} comp_num;

__device__ void print_comp(comp_num* c);
__device__ comp_num add(comp_num c1, comp_num c2, comp_num* r);
__device__ comp_num mult(comp_num c1, comp_num c2, comp_num* r);
__device__ float absSq(comp_num c);

__device__ float scale_between(float num, float min, float max, float newMin, float newMax);


extern "C"
__global__ void mandelbrot(int n, int maxIt, int w, int h, float minX, float minY, float maxX, float maxY, float offsetX, float offsetY, float* it) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < n)
    {
        int x = i % w;
        int y = (i - x) / w;
        comp_num c, p;
        c.r = 0;
        c.i = 0;
        p.r = scale_between(x + offsetX, 0, 800, minX, maxX);
        p.i = scale_between(y + offsetY, 0, 800, minY, maxY);

        int n = 0;
        while(n < maxIt && absSq(c) < 4)
        {
            comp_num new_c;
            mult(c, c, &new_c);
            c = new_c;
            add(c, p, &new_c);
            c = new_c;

            n++;
        }

        it[i] = n + 1 - logf(log2f(sqrtf(absSq(c))));
    }
}

__device__ void print_comp(comp_num* c)
{
    printf("%.2f + %.2fi\n", c->r, c->i);
}

__device__ comp_num add(comp_num c1, comp_num c2, comp_num* r)
{
    r->r = c1.r + c2.r;
    r->i = c1.i + c2.i;
}

__device__ comp_num mult(comp_num c1, comp_num c2, comp_num* r)
{
    r->r = c1.r * c2.r - c1.i * c2.i;
    r->i = c1.r * c2.i + c1.i * c2.r;
}

__device__ float absSq(comp_num c)
{
    return c.r * c.r + c.i * c.i;
}

__device__ float scale_between(float num, float min, float max, float newMin, float newMax)
{
    return ((newMax - newMin)*(num - min))/(max - min) + newMin;
}